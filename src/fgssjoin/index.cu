#include "hip/hip_runtime.h"
#include "index.cuh"
#include "scan.cuh"



/*
 *  DOCUMENTATION
 */
__host__ 
Index inverted_index (sets_t* sets, float threshold, char verbose) 
{
	unsigned long t0, t1;

    if (verbose) {
        fprintf(stderr, "* Building inverted index...\n");
        t0 = ut_get_time_in_microseconds();
    }

    Entry* d_entries = create_midpref_entries(sets, sets->num_midpref_tokens, 0, sets->num_sets);

    int num_sets    = (int) sets->num_sets;
    int num_terms   = (int) sets->num_terms;
    int num_entries = (int) sets->num_midpref_tokens;
    Entry *d_lists;
    int *d_count, *d_index;


    gpu(hipMalloc(&d_lists, num_entries * sizeof(Entry)));
    gpu(hipMalloc(&d_index, num_terms * sizeof(int)));
    gpu(hipMalloc(&d_count, num_terms * sizeof(int)));   
    gpu(hipMemset(d_count, 0, num_terms * sizeof(int)));


    dim3 grid, block;
    get_grid_config(grid, block);
    df_count_kernel <<<grid, block>>> (d_entries, d_count, num_entries);     
    exclusive_scan <int> (d_count, d_index, num_terms);
    inverted_index_kernel <<<grid, block>>> (d_entries, d_lists, d_index, num_entries);
    gpu(hipDeviceSynchronize());


    Index index = Index(d_lists, d_index, d_count, num_sets, num_terms, num_entries);
    gpu(hipFree(d_entries));

    if (verbose) {
        t1 = ut_get_time_in_microseconds();
        fprintf(stderr, "  - Done. It took %g ms.\n", ut_interval_in_miliseconds(t0,t1));
    }
    
    return index;
}



/*
 *  DOCUMENTATION
 */
__host__ 
void print_inverted_index (Index invindex, sets_t* sets)
{
	int num_terms   = invindex.num_terms;
	int num_entries = invindex.num_entries;
	int *index      = (int*) malloc(num_terms * sizeof(int));
	int *count      = (int*) malloc(num_terms * sizeof(int));
	Entry *entries  = (Entry*) malloc(num_entries * sizeof(Entry));

	gpu(hipMemcpy(index, invindex.d_index, num_terms * sizeof(int), hipMemcpyDeviceToHost));
	gpu(hipMemcpy(count, invindex.d_count, num_terms * sizeof(int), hipMemcpyDeviceToHost));
	gpu(hipMemcpy(entries, invindex.d_lists, num_entries * sizeof(Entry), hipMemcpyDeviceToHost));

	for (int i = 0; i < num_terms; i++) {
		int pos = i == 0 ? 0 : index[i-1];
		int len = count[i];
		fprintf(stderr, "term: (%d)\n", entries[pos].term_id);
		for (int j = 0; j < len; j++)
			fprintf(stderr, "(%u) ", sets->id[entries[pos + j].set_id]);
		fprintf(stderr, "\n\n");
	}

	free(index);
	free(count);
	free(index);
}



/*
 *  DOCUMENTATION
 */
__host__
Entry* create_midpref_entries (
    sets_t* sets, unsigned int n_entries, 
    unsigned int offset, unsigned int block_size
)
{
    dim3 grid, block;
    get_grid_config(grid, block);
    
    Entry* d_entries;
    gpu(hipMalloc(&d_entries, n_entries * sizeof(Entry)));
    
    midpref_kernel <<<grid, block>>> (d_entries, 
    	sets->d_midpref_pos, sets->d_midpref_len, sets->d_midpref_tokens, 
        offset, block_size
	);
    gpu(hipDeviceSynchronize());

    return d_entries;
}



/*
 *  DOCUMENTATION
 */
__global__
void midpref_kernel (
    Entry* entries, unsigned int* midpref_pos, unsigned int* midpref_len,
    unsigned int* midpref_tokens, unsigned int offset, unsigned int n_sets
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < n_sets; i += blockDim.x * gridDim.x)
    {
        int x = i + offset;
        for (unsigned j = 0; j < midpref_len[x]; j++) 
        {
            int set = x;
            int idx = midpref_pos[x] + j;
            int token = midpref_tokens[idx];
            entries[idx].set_id = set;
            entries[idx].term_id = token;
            entries[idx].pos = (int) j;
        }
    }
}



/*
 *  DOCUMENTATION
 */
__global__ 
void df_count_kernel (Entry *entries, int *count, int n) 
{
    int block_size = n / gridDim.x + (n % gridDim.x == 0 ? 0 : 1);
    int offset = block_size * (blockIdx.x);
    int lim = offset + block_size;
    if (lim >= n) lim = n;
    int size = lim - offset;

    entries += offset;

    for (int i = threadIdx.x; i < size; i+= blockDim.x) 
    {
        int term_id = entries[i].term_id;
        atomicAdd(count + term_id, 1);
    }
}



/*
 *  DOCUMENTATION
 */
__global__ 
void inverted_index_kernel(Entry *entries, Entry *lists, int *index, int n) 
{  
    int block_size = n / gridDim.x + (n % gridDim.x == 0 ? 0 : 1);
    int offset = block_size * (blockIdx.x);
    int lim = offset + block_size;
    if (lim >= n) lim = n;
    int size = lim - offset;

    entries += offset;

    for (int i = threadIdx.x; i < size; i+= blockDim.x) 
    {
        Entry entry = entries[i];
        int pos = atomicAdd(index + entry.term_id, 1);
        lists[pos] = entry;
    }
}